#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
 #define VECTORSIZE 12
 __global__ void vector_addition(int *A, int *B, int *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int sum = 0;
        for (int j = 0; j < n; j++) {
            sum += A[i * n + j] * B[j];
        }
        C[i] = sum;
    }
}
 int main() {
    int *A, *B, *C;
    int n = VECTORSIZE * VECTORSIZE;
    size_t size = n * sizeof(int);
     A = (int *)malloc(size);
    B = (int *)malloc(VECTORSIZE * sizeof(int));
    C = (int *)malloc(VECTORSIZE * sizeof(int));
     // Initialize data
    for (int i = 0; i < n; i++) {
        A[i] = 1;
    }
    for (int i = 0; i < VECTORSIZE; i++) {
        B[i] = 1;
    }
     // Allocate device memory
    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, VECTORSIZE * sizeof(int));
    hipMalloc((void **)&d_C, VECTORSIZE * sizeof(int));
     // Copy data to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, VECTORSIZE * sizeof(int), hipMemcpyHostToDevice);
     // Launch kernel
    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    struct timeval start_time, stop_time;
    gettimeofday(&start_time, NULL);
    vector_addition<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, VECTORSIZE);
    hipDeviceSynchronize();
    gettimeofday(&stop_time, NULL);
     // Copy result back to host memory
    hipMemcpy(C, d_C, VECTORSIZE * sizeof(int), hipMemcpyDeviceToHost);
     // Print result and execution time
    printf("\nVector addition output: \n");
    for (int i = 0; i < VECTORSIZE; i++) {
        printf("\t%d", C[i]);
    }
    double exe_time = (stop_time.tv_sec + (stop_time.tv_usec / 1000000.0)) - (start_time.tv_sec + (start_time.tv_usec / 1000000.0));
    printf("\n Execution time is = %lf seconds\n", exe_time);
     // Free memory
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
     return 0;
}