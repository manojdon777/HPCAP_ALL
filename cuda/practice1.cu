
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__device__ int calc_square(int val)
{
	int val_square;
	val_square = val*val;
	return val_square;
}
__global__ void arr_init(int *a){
	int myid = threadIdx.x;
	a[myid] = myid;
}
__global__ void arr_square(int *a){
	int myid = threadIdx.x;
	a[myid] = calc_square(a[myid]);
}
int main()
{
	int size = 400 * sizeof(int);
	int a[400], aa[400];
	int *ad, *aad;
	int i=0;
	
	hipMalloc(&ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMalloc(&aad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	dim3 DimGrid(1, 1);
	dim3 DimBlock(400, 1);
	arr_init<<< DimGrid, DimBlock >>>(ad);
	hipMemcpy(a,ad,size, hipMemcpyDeviceToHost);
	arr_square<<< DimGrid,DimBlock >>>(aad);
	hipMemcpy(aa, aad, size, hipMemcpyDeviceToHost);
	// Add two vectors
	// for(i=0; i<400; i++ )
	// {
	// 	aa[i] = calc_square(a[i]);
	// }

	// print the output
	for(i=0; i<400; i++ )
	{
		printf("\t%d",aa[i]);
	}
}
