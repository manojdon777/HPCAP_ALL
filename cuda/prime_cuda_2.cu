
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define N 1000000

#define THDS_PER_BLK 256
__global__ void sum_reduce(int *arr, int *sum)
{
    int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	int range = THDS_PER_BLK/2;
	__shared__ int tmp[THDS_PER_BLK];
	tmp[threadIdx.x] = 0;
    if(myid<N)
    {
        tmp[threadIdx.x] = arr[myid];
        __syncthreads();
        while(range>0)
        {
            if(threadIdx.x < range)
            {
                tmp[threadIdx.x] += tmp[threadIdx.x + range];
            }
            range = range /2;
            __syncthreads(); 
        }
        if(threadIdx.x == 0)
        {
            sum[blockIdx.x] = tmp[threadIdx.x]; 
        }
    }
}

__global__ void prime_calc(int *sum)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	int i,j, flag;
	flag = 1;
	i = myid;
	if((myid<N) && (myid>2))
	{
		for(j=2;j<i;j++)	
	    {
		    if((i%j) == 0)
		    {
			    flag = 0;
			    break;
		    }
	    }
	    sum[myid] = flag;
    }
}

int main()
{
	int *sum, *sum_d, *sum_small_d;
	int i=0;
	int total = 1;
	double exe_time;
	struct timeval stop_time, start_time;
    
    int thds_per_block = THDS_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;
	
	sum = (int *)malloc(N*sizeof(int));
	
	hipMalloc(&sum_d, N*sizeof(int));
	hipMalloc(&sum_small_d, num_blocks*sizeof(int));
	
	gettimeofday(&start_time, NULL);	
	
	prime_calc<<< num_blocks,thds_per_block >>>(sum_d);
	hipDeviceSynchronize();
	
	sum_reduce<<< num_blocks,thds_per_block >>>(sum_d, sum_small_d);
	hipMemcpy(sum, sum_small_d, num_blocks*sizeof(int), hipMemcpyDeviceToHost);
	
	for(i=0; i<num_blocks; i++)
	{
        total += sum[i];
    }
    
    gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	    
    printf("\n prime numbers = %d and exe_time = %lf\n", total, exe_time);	
    hipFree(sum_d); 
    free(sum);
}
