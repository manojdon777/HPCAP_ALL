#include<stdio.h>
#include<stdlib.h>
#define N 900000000

_global__ void arradd(double *sum)
{
        int myid = blockIdx.x*blockDim.x + threadIdx.x;
        step = 1.0/(double)N;
        x = (myid)*step;
        sum[myid] =4.0/(1.0+x*x);
}

int main(){
        double *sum, ;
        step = 1.0/(double)N;
        hipMalloc(&pi, size);
        hipMemcpy(&stepd, step, size, hipMemcpyHostToDevice);
        dim3   DimGrid(1, 1);
        dim3   DimBlock(N, 1);
        arradd<<< DimGrid,DimBlock >>>(x, step, pid);
        hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);



        pi = step*sum;
        if(myid==0)
                printf("Pi = %f and time taken = %f\n", pi, total_time);
        return 0;
}