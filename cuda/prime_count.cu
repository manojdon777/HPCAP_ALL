
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<omp.h>
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/

#define N 1000
__global__ void arradd(int *countd)
{
	// int myid = threadIdx.x;
	int myid = blockIdx.x * blockDim.x + threadIdx.x;
	int flag = 1;
	if(myid>2 && myid<N)
	{
		flag = 0;
		for(int j=2;j<myid;j++)
		{
			if((myid%j) == 0)
			{
				countd[myid] = 0;
				break;
			}
		}
		if(flag == 0)
		{
			countd[myid] = 1;
		}
	}
}
int main()
{
	int *countd, count[N];
	int counter = 0;
	hipMalloc(&countd, N*sizeof(int));	
	// cudaMemcpy(countd, count, N*sizeof(int), cudaMemcpyHostToDevice);
	dim3   DimGrid(1, 1);
	dim3   DimBlock(N, 1);
	arradd<<< DimGrid,DimBlock >>>(countd);
	hipMemcpy(count, countd, sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i<N; i++)
		if(count[i] == 1)
			counter++;
	printf("Number of primes: %d.\n", counter);
	return 0;
}