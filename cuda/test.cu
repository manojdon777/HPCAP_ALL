
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.x;

	pd[myid] = md[myid] + nd[myid];
}
#define ARRSIZE 100000
int main()
{
	int size = ARRSIZE * sizeof(int);
	int m[ARRSIZE], n[ARRSIZE], p[ARRSIZE],*md, *nd,*pd;
	int i=0;  
	for(i=0; i<ARRSIZE; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}  
	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);
	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);
	hipMalloc(&pd, size);
	dim3   DimGrid(1, 1);
	dim3   DimBlock(ARRSIZE, 1);
	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);
	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	for(i=0; i<ARRSIZE; i++ )
	{
		printf("\t%d",p[i]);
	}
	printf("\n");
	hipFree(md);
	hipFree(nd);
	hipFree(pd);
}